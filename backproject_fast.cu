#include "hip/hip_runtime.h"
/* backprojects */
#include <math.h>
#include <matrix.h>
#include <mex.h>

#define MAX_LASERS 100
#define MAX_NX 20000

#define dist(v1, v2) \
	sqrt(((v1)[0]-(v2)[0])*((v1)[0]-(v2)[0])\
		+((v1)[1]-(v2)[1])*((v1)[1]-(v2)[1])\
		+((v1)[2]-(v2)[2])*((v1)[2]-(v2)[2]))

#define VOXEL_CHUNCK 32768
#define LASER_CHUNCK 32
#define CAM_CHUNCK 32

void __global__ calChunck(double *d1l, double *laserpos, double *voxels,
	double *cpos, double *d4l, double shift, double tpp, int nlasers, int nx,
	int nt, int nvoxels, double *sI, double intensity_correction, double *output)
{
	__shared__ double sum[LASER_CHUNCK * CAM_CHUNCK];
	int voxelIdx = threadIdx.x * CAM_CHUNCK + threadIdx.y;
	if (blockIdx.x >= nvoxels || threadIdx.x >= nlasers || threadIdx.y >= nx)
		return;
	double d2 = dist(&laserpos[threadIdx.x*3], &voxels[blockIdx.x*3]);
	double d3 = dist(&voxels[blockIdx.x*3], &cpos[threadIdx.y*3]);
	double d = d1l[threadIdx.x] + d2 + d3 + d4l[threadIdx.y];
	int tindex = (d-(shift))/(tpp) + 0.5;
	if ((tindex>=0) && (tindex<nt))
	{                          
		int index = threadIdx.x*nx*nt + tindex +threadIdx.y*nt;
		sum[voxelIdx] = sI[index%50000] * (intensity_correction);
	} else {
		sum[voxelIdx] = 0;
	}
	__syncthreads();
	if (threadIdx.x != 0 || threadIdx.y != 0)
		return;
	double result = 0;
	for (int i = 0; i < nlasers; i++) {
		for (int j = 0; j < nx; j++) {
			result += sum[i*CAM_CHUNCK+j];
		}
	}
	output[blockIdx.x] += result;
}

//Load Variables
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
/* Macros for the ouput and input arguments */
#define xsI prhs[0]
#define xlaserpos prhs[1]
#define xvoxels prhs[2]
#define xcpos prhs[3]
#define xlcop prhs[4]
#define xccop prhs[5]
#define xtpp prhs[6]
#define xshift prhs[7]
#define xlasernorm prhs[8]
#define xcameranorm prhs[9]

#define xoutput plhs[0]

hipError_t rc;

double *sI = mxGetPr(xsI);
double *laserpos = mxGetPr(xlaserpos);
double *voxels = mxGetPr(xvoxels);
double *cpos = mxGetPr(xcpos);
double *geo_laser_cop = mxGetPr(xlcop);
double *geo_camera_cop = mxGetPr(xccop);
double *cameranormal = mxGetPr(xcameranorm);
double *lasernormal = mxGetPr(xlasernorm);

double tpp = *((double *)mxGetData(xtpp)); 
double shift = *((double *)mxGetData(xshift));

int msI = mxGetM(xsI);
int nlasers = mxGetN(xsI);
int nx = mxGetN(xcpos);
int nt = mxGetM(xsI)/nx;
int nvoxels = mxGetN(xvoxels);

double *d_sI, *d_laserpos, *d_voxels, *d_cpos;

rc = hipMalloc((void **)&d_sI, sizeof(double)*msI*nlasers);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
rc = hipMemcpy(d_sI, sI, sizeof(double)*msI*nlasers, hipMemcpyHostToDevice);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));

rc = hipMalloc((void **)&d_laserpos, sizeof(double)*3*nlasers);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
rc = hipMemcpy(d_laserpos, laserpos, sizeof(double)*3*nlasers, hipMemcpyHostToDevice);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));

rc = hipMalloc((void **)&d_cpos, sizeof(double)*3*nx);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
rc = hipMemcpy(d_cpos, cpos, sizeof(double)*3*nx, hipMemcpyHostToDevice);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));

int x = 0,p=0;
int tpos = 0;

mexPrintf("nlasers: %d\n", nlasers);
mexPrintf("nvoxels: %d\n", nvoxels);
mexPrintf("nx: %d\n", nx);
mexPrintf("nt: %d\n", nt);
mexPrintf("tpp: %f\n", tpp);
mexPrintf("shift: %f\n", shift);
mexPrintf("First voxel: %f %f %f\n", voxels[0], voxels[1], voxels[2]);
mexPrintf("Laser cop: %f %f %f\n", geo_laser_cop[0], geo_laser_cop[1], geo_laser_cop[2]);


//Start Backproject
mxArray *out_array = xoutput = mxCreateDoubleMatrix(nvoxels,1,mxREAL);

double *output = mxGetPr(out_array);
double* d1l=new double[nlasers];
double* d4l=new double[nx];


for( tpos = 0;tpos<nlasers;tpos++) {
  d1l[tpos] = dist(geo_laser_cop,&laserpos[tpos*3]);
//  mexPrintf("geo_laser_cop [%f %f %f]\n", geo_laser_cop[0], geo_laser_cop[1], geo_laser_cop[2]);
}

for (x=0;x<nx;x++) {
  d4l[x] = dist(&cpos[x*3],geo_camera_cop);
}

double *d_output;
double *d_d1l, *d_d4l;

rc = hipMalloc((void **)&d_d1l, sizeof(double)*nlasers);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
rc = hipMemcpy(d_d1l, d1l, sizeof(double)*nlasers, hipMemcpyHostToDevice);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));

rc = hipMalloc((void **)&d_d4l, sizeof(double)*nx);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
rc = hipMemcpy(d_d4l, d4l, sizeof(double)*nx, hipMemcpyHostToDevice);
if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));

for (p = 0; p < nvoxels; p += VOXEL_CHUNCK) {
	mexPrintf("%d percent done\n", (p / (nvoxels / 10)) * 10);
	// mexPrintf("%d %d\n",p, nvoxels);
	mexEvalString("pause(.001);"); // to dump string.
	
	int vchunck = min(VOXEL_CHUNCK, nvoxels - p);
	
	rc = hipMalloc((void **)&d_voxels, sizeof(double)*3*vchunck);
	if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
	rc = hipMemcpy(d_voxels, &voxels[3*p], sizeof(double)*3*vchunck, hipMemcpyHostToDevice);
	if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
	
	rc = hipMalloc((void **)&d_output, sizeof(double)*vchunck);
	if (rc != hipSuccess) printf(	"ERROR ON CUDA: %s\n", hipGetErrorString(rc));
	rc = hipMemset(d_output, 0, sizeof(double)*vchunck);
	if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
	
	for (tpos = 0; tpos < nlasers; tpos += LASER_CHUNCK) {
		int lchunck = min(LASER_CHUNCK, nlasers - tpos);
		
		for (x = 0; x < nx; x += CAM_CHUNCK) {
			int xchunck = min(CAM_CHUNCK, nx - x);
	
			calChunck<<<VOXEL_CHUNCK, dim3(LASER_CHUNCK, CAM_CHUNCK)>>>(&d_d1l[tpos], &d_laserpos[3*tpos],
				d_voxels, &d_cpos[3*x], &d_d4l[x], shift, tpp, lchunck, xchunck, nt, vchunck, d_sI, 1.0, d_output);
		}
	}
	
	rc = hipMemcpy(&output[p], d_output, sizeof(double)*vchunck, hipMemcpyDeviceToHost);
	if (rc != hipSuccess) printf("ERROR ON CUDA: %s\n", hipGetErrorString(rc));
}

hipFree(d_d1l);
hipFree(d_d4l);
hipFree(d_sI);
hipFree(d_voxels);
hipFree(d_laserpos);
hipFree(d_cpos);
hipFree(d_output);

/*
for(p=0;p<nvoxels;p++)
{
	break;
  if (p % (nvoxels / 10) == 0)
  {
      mexPrintf("%d percent done\n", (p / (nvoxels / 10)) * 10);
      // mexPrintf("%d %d\n",p, nvoxels);
      mexEvalString("pause(.001);"); // to dump string.
  }
    
  double thesum=0;
  double * voxel1 = &voxels[p*3];
  
  for( tpos = 0;tpos<nlasers;tpos++) {
      double * laserpos1 = &laserpos[tpos*3];
      double * lasernorm1 = &lasernormal[tpos*3];
      double d1 = d1l[tpos];
      double d2 = distance2(laserpos1,voxel1);
      
      for (x=0;x<nx;x++) {
          double * cpos1 = &cpos[x*3];
          double * cameranorm1 = &cameranormal[x*3];
          double d3 = distance2(voxel1,cpos1);
          double d4 = d4l[x];
          double d=d1+d2+d3+d4;
          
          //mexPrintf("d1: %f, d2: %f d3: %f, d4: %f\n", d1, d2, d3, d4);
          
          double vlv[3];
          double vcv[3];
          vlv[0]= (voxel1[0]-laserpos1[0])/d2;
          vlv[1]= (voxel1[1]-laserpos1[1])/d2;
          vlv[2]= (voxel1[2]-laserpos1[2])/d2;
          double dotlv = vlv[0]*lasernorm1[0]+vlv[1]*lasernorm1[1]+vlv[2]*lasernorm1[2];
//           mexPrintf("vlv: %f\n" , vlv[2]);
//           mexPrintf("laser normal: %f %f %f\n", lasernormal[0], lasernormal[1], lasernormal[2]);
          
          vcv[0]= (voxel1[0]-cpos1[0])/d3;
          vcv[1]= (voxel1[1]-cpos1[1])/d3;
          vcv[2]= (voxel1[2]-cpos1[2])/d3;
          double dotcv = vcv[0]*cameranorm1[0]+vcv[1]*cameranorm1[1]+vcv[2]*cameranorm1[2];
//           mexPrintf("vcv: %f\n", vcv[2]);
//           mexPrintf("camera normal: %f %f %f\n", cameranormal[0], cameranormal[1], cameranormal[2]);
          
          double intensity_correction = 1.0;          
//           intensity_correction =  sqrt(d2*d3);
//           intensity_correction =  d2*d3;
          
          int tindex = round((d-(shift))/tpp);
                  
          int index = 0;
          double tol=0.3;
          if(voxel1[0]>-tol && voxel1[0]<tol && voxel1[1]>46-tol && voxel1[1]<46+tol && voxel1[2]>-40-tol && voxel1[2]<-40+tol) {
                mexPrintf("---------------------------------------------------------\n");
                mexPrintf("Voxel 4000: [%f %f %f]\n", voxel1[0], voxel1[1], voxel1[2]);
                mexPrintf("cpos: [%f %f %f]\n", cpos1[0], cpos1[1], cpos1[2]);
                mexPrintf("lpos: [%f %f %f]\n", laserpos1[0], laserpos1[1], laserpos1[2]);
                mexPrintf("d1 %f, d2 %f, d3 %f, d4 %f\n", d1, d2, d3, d4);
                mexPrintf("INDEX: %d\n", tindex);
                mexPrintf("d %f\n", d);
                }
                      
          if ((tindex>=0) && (tindex<nt) && (dotlv>0) && (dotcv>0))
          //if ((tindex>=0) && (tindex<nt))
          {                          
              index = tpos*nx*nt + tindex +x*nt;                            
              thesum = thesum + sI[index] *intensity_correction;
          }

          /*
          double tindexd = ((d-(shift))/tpp);
          int tindexl = floor(tindexd), tindexu=ceil(tindexd);  
         
          if ((tindexl>=0) && (tindexu<nt))
          {            
              double w = tindexd-tindexl;
              int indexl = tpos*nx*nt + tindexl +x*nt, indexu = tpos*nx*nt + tindexu +x*nt;                                       
              thesum = thesum + ( (1-w) * sI[indexl] + w * sI[indexu])*intensity_correction;
              //sIout[index] = -1;
          }*//*
         
           
          if (x == 250 && p == 400 && tpos==0)

          {
              mexPrintf("tindex: %d\n", tindex);
              mexPrintf("d1: %f\n", d1);
              mexPrintf("d2: %f\n", d2);
              mexPrintf("d3: %f\n", d3);
              mexPrintf("d4: %f\n", d4);
              mexPrintf("Laser pos: %f %f %f\n", laserpos1[0], laserpos1[1], laserpos1[2]);
              mexPrintf("Point pos: %f %f %f\n", voxel1[0], voxel1[1], voxel1[2]);
              mexPrintf("Cam pos: %f %f %f\n", cpos1[0], cpos1[1], cpos1[2]);
              mexPrintf("x: %d\n", x);
              mexPrintf("pixel index: %d\n",  tindex +x*nt);
              //mexPrintf("pixel value: %f\n", sI[tpos*nx*nt + tindex +x*nt]);
              mexPrintf("pixel value: %f\n", sI[index]);
          }
          
      }
  }
  output[p] = thesum; 
}*/
mexPrintf("100 percent done\n");
delete[] d1l;
delete[] d4l;
}
